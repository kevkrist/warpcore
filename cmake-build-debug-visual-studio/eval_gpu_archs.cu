

#include <hip/hip_runtime.h>
#include <cstdio>
#include <set>
#include <string>
using namespace std;
int main(int argc, char** argv) {
  set<string> archs;
  int nDevices;
  if((hipGetDeviceCount(&nDevices) == hipSuccess) && (nDevices > 0)) {
    for(int dev=0;dev<nDevices;++dev) {
      char buff[32];
      hipDeviceProp_t prop;
      if(hipGetDeviceProperties(&prop, dev) != hipSuccess) continue;
      sprintf(buff, "%d%d", prop.major, prop.minor);
      archs.insert(buff);
    }
  }
  if(archs.empty()) {
    printf("ALL");
  } else {
    bool first = true;
    for(const auto& arch : archs) {
      printf(first? "%s" : ";%s", arch.c_str());
      first = false;
    }
  }
  printf("\n");
  return 0;
}
